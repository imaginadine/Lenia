#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <math.h>
#include <time.h>

// OpenGL Graphics includes
#include <GL/glew.h>
#ifdef _WIN32
#include <GL/wglew.h>
#endif
#if defined(__APPLE__) || defined(__MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA runtime
// CUDA utilities and system includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define SCREEN_X 512
#define SCREEN_Y 512
#define FPS_UPDATE 500
#define TITLE "Lenia"

#define CPU_MODE 1
#define GPU_MODE 2
#define OPENGL_GPU_MODE 3

int width_grid = 256;
int height_grid = 256;

int block_dim_x = 16;
int block_dim_y = 16;
int grid_dim_x = (width_grid + block_dim_x - 1) / block_dim_x;
int grid_dim_y = (height_grid + block_dim_y - 1) / block_dim_y;

float4* d_grid1, * d_grid2;
bool tab_1_used = true;

GLuint imageTex;
GLuint imageBuffer;

GLuint glBuffer;
GLuint glTex;
struct hipGraphicsResource* cuBuffer;

float* debug;

/* Globals */
float scale = 0.003f;
float mx = 0.f;
float my = 0.f;
int mode = CPU_MODE;
int frame = 0;
int timebase = 0;

float4* pixels2;
int size = width_grid * height_grid * sizeof(float4);

#define INF 2e10f

unsigned long long seed = time(NULL); // or any other unique seed value

// Lenia parameters

#define R 10
#define T 10
#define mu 0.15f
#define omega 0.016f
#define alpha 4
#define B 1 // rank for the pics

float4* lenia_pixels;

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors
(hipError_t err, const char* file, const int line)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString(err));
		system("pause");
		exit(1);
	}
}

float4* zeroPixels() {
	float4* p = (float4*)malloc(size);
	for (int i = 0; i < width_grid * height_grid;i++) {
		p[i].x = 0.0f;
		p[i].y = 0.0f;
		p[i].z = 0.0f;
		p[i].w = 1.0f;
	}
	return p;
}

float4* randomPixels() {
	float4* p = (float4*)malloc(size);
	for (int i = 0; i < width_grid * height_grid; i++) {
		if (i % (10 * width_grid) == 0) srand(i);
		float random_value = (float)rand() / RAND_MAX; // Generate a random float between 0 and 1
		p[i].x = 0.0f;
		p[i].y = random_value; 
		p[i].z = 0.0f;
		p[i].w = 1.0f;
	}
	return p;
}

float4* randomPixelsCenter() {
	float4* p = (float4*)malloc(size);
	for (int i = 3 * height_grid/8; i < 5*height_grid/8; i++) {
		for (int j = 3 * width_grid / 8; j < 5 * width_grid / 8; j++) {
			int index = i * width_grid + j;

			if (index % (10 * width_grid) == 0) srand(index);
			float random_value = (float)rand() / RAND_MAX; // Generate a random float between 0 and 1
			p[index].x = 0.0f;
			p[index].y = random_value;
			p[index].z = 0.0f;
			p[index].w = 1.0f;
		}
		
	}
	return p;
}

float4* orbium()
{	
	float4* p = zeroPixels();
	p[6].y = 0.1f;
	p[7].y = 0.14f;
	p[8].y = 0.1f;
	p[11].y = 0.03f;
	p[12].y = 0.03f;
	p[15].y = 0.3f;
	return p;
}

void initCPU()
{
	time_t t;

	// Intializes random number generator
	srand((unsigned)time(&t));

	lenia_pixels = randomPixelsCenter();
	pixels2 = zeroPixels();

}

void cleanCPU()
{
	free(lenia_pixels);
	free(pixels2);
}


void initGPU()
{
	time_t t;

	// Intializes random number generator
	srand((unsigned)time(&t));

	lenia_pixels = randomPixels();
	pixels2 = zeroPixels();
	checkCudaErrors(hipMalloc((void**)&d_grid1, size));
	checkCudaErrors(hipMalloc((void**)&d_grid2, size));

}

void cleanGPU()
{
	free(lenia_pixels);
	free(pixels2);
	hipFree(d_grid1);
	hipFree(d_grid2);
}

// ------------------------- LENIA -------------------------------------------------------------------------------------

/*
	exponential core function
	pre-condition : r between 0 and 1
	post-condition : return value between O and 1
*/
__host__ __device__ float kernel_core_exp(float r)
{
	return exp(alpha - (alpha / (4 * r * (1 - r))));
}

__host__ __device__ float growth_function_exp(float u)
{
	float dividende = (2.0f * omega * omega);
	return 2.0f * exp(-((u - mu) * (u - mu)) / dividende) - 1.0f;
}

/*
	kernel shell
	pre-conditions : r between 0 and 1 ; beta in [0;1] dim B
	post-condition : return value between 0 and 1
*/
/*float kernel_shell(float r, float beta[B])
{
	int index = static_cast<int>(B * r);
	double fraction = B * r - index;

	double peak_height = beta[index];

	return peak_height * kernel_core_exp(fraction);
}*/

/*
	Normalization of the kernel
	pre-condition : n in the neighbourhood, at the indexes i and j
	post-condition : 
*/
__host__ __device__ float normalized_kernel(int x, int y, float beta[B], int width_grid, int height_grid)
{
	float norm_n = sqrt(x*x + y*y);
	float Ks_val = kernel_core_exp(norm_n); //kernel_shell(norm_n, beta)

	float sum = 0.0f;
	// sum of the Ks of the neighbourhood (ie where norm(x) <= R)
	for (int i = x-R; i <= x+R;i++) {
		for (int j = y-R; j <= y+R;j++) {
			if (i != x || j != y) {
				int wrappedI = (i + height_grid) % height_grid;
				int wrappedJ = (j + width_grid) % width_grid;
				sum += kernel_core_exp(sqrt(wrappedI * wrappedI + wrappedJ * wrappedJ));//kernel_shell(sqrt(wrappedI * wrappedI + wrappedJ * wrappedJ), beta);
			}
		}
	}

	return Ks_val / sum;
}

/*
	Potential distribution U_t(x) - Local rule
	Pre-condition : in the grid at indexes x and y
	Post-condition : return value between 0 and 1
*/
__host__ float potential_distribution(int x, int y, int width, int height)
{
	float sum = 0.0f;
	float beta[B];
	beta[0] = 1.0f;

	float n_kernel = normalized_kernel(x, y, beta, width, height);

	for (int i = x - R; i <= x + R;i++) {
		for (int j = y - R; j <= y + R;j++) {
			if (i != x || j != y) {
				// calculate the wrapped index
				int wrappedI = (i + height_grid) % height_grid;
				int wrappedJ = (j + width_grid) % width_grid;

				sum += n_kernel * lenia_pixels[wrappedI * width_grid + wrappedJ].y;
			}
		}
	}

	return sum;
}



void lenia_basic_CPU()
{
	// for each pixel
	for (int i = 0; i < height_grid; i++) {
		for (int j = 0; j < width_grid; j++) {
			float c_t = lenia_pixels[i * width_grid + j].y; // field C at time step t
			//printf("c_t = %f - ", c_t);
			float c_tdt; // field C at time step t + delta(t) ; (delta (t) = 1/T)

			// 1st step : convolution operation, multiplication with the kernel
			float u_t = potential_distribution(i, j, width_grid, height_grid);
			//printf("u_t = %f - ", u_t);
			// 2nd step : growth mapping
			float g_t = growth_function_exp(u_t);
			//printf("g_t = %f - ", g_t);
			// 3rd step : add the growth to the existing value
			float dt = 1.0f / float(T);
			c_tdt = c_t + dt * g_t;
			//printf("c_t step 3 = %f - ", c_tdt);
			// 4th step : clip the result to be in range from 0 to 1
			if (c_tdt < 0.0f) c_tdt = 0.0f;
			if (c_tdt > 1.0f) c_tdt = 1.0f;

			//printf("c_tdt = %f\n", c_tdt);

			// assign the value to a temporary table
			pixels2[i * width_grid + j].y = c_tdt;
		}
	}
}


__global__ void lenia (float4* d_grid_old, float4* d_grid_new, int width, int height)
{
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;

	if (indexX < width && indexY < height) {
		int index = indexY * width + indexX;

		float c_t = d_grid_old[index].y; // field C at time step t
		float c_tdt; // field C at time step t + delta(t) ; (delta (t) = 1/T)

		// 1st step : convolution operation, multiplication with the kernel

		float sum = 0.0f;
		float beta[B];
		beta[0] = 1.0f;

		float n_kernel = normalized_kernel(indexX, indexY, beta, width, height);

		for (int i = indexX - R; i <= indexX + R;i++) {
			for (int j = indexY - R; j <= indexY + R;j++) {
				if (i != indexX || j != indexY) {
					// calculate the wrapped index
					int wrappedI = (i + height) % height;
					int wrappedJ = (j + width) % width;

					sum += n_kernel * d_grid_old[wrappedI * width + wrappedJ].y;
				}
			}
		}
		float u_t = sum;

		// 2nd step : growth mapping
		float g_t = growth_function_exp(u_t);

		// 3rd step : add the growth to the existing value
		float dt = 1.0f / float(T);
		c_tdt = c_t + dt * g_t;

		// 4th step : clip the result to be in range from 0 to 1
		if (c_tdt < 0.0f) c_tdt = 0.0f;
		if (c_tdt > 1.0f) c_tdt = 1.0f;

		// assign the value
		d_grid_new[index].y = c_tdt;
	}
}


void lenia_basic_GPU()
{
	dim3 dimBlock(block_dim_x, block_dim_y);
	dim3 dimGrid(grid_dim_x, grid_dim_y);

	hipError_t err;

	if (tab_1_used)
	{
		// send lenia pixels to device
		checkCudaErrors(hipMemcpy(d_grid1, lenia_pixels, size, hipMemcpyHostToDevice));

		// do treatments
		lenia << <dimGrid, dimBlock >> > (d_grid1, d_grid2, width_grid, height_grid);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Error: % s\n", hipGetErrorString(err));
		}

		// fetch grid 2
		checkCudaErrors(hipMemcpy(pixels2, d_grid2, size, hipMemcpyDeviceToHost));
	}
	else {
		// send lenia pixels to device
		checkCudaErrors(hipMemcpy(d_grid2, pixels2, size, hipMemcpyHostToDevice));

		// do treatments
		lenia << <dimGrid, dimBlock >> > (d_grid2, d_grid1, width_grid, height_grid);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Error: % s\n", hipGetErrorString(err));
		}

		// fetch grid 1
		checkCudaErrors(hipMemcpy(lenia_pixels, d_grid1, size, hipMemcpyDeviceToHost));
	}

}


void calculate() {
	frame++;
	int timecur = glutGet(GLUT_ELAPSED_TIME);

	if (timecur - timebase > FPS_UPDATE) {
		char t[200];
		char* m = "";
		switch (mode)
		{
		case CPU_MODE: m = "CPU mode"; break;
		case GPU_MODE: m = "GPU mode"; break;
		//case OPENGL_GPU_MODE: m = "GPU mode - OpenGL interoperability"; break;
		}
		sprintf(t, "%s:  %s, %.2f FPS", TITLE, m, frame * 1000 / (float)(timecur - timebase));
		glutSetWindowTitle(t);
		timebase = timecur;
		frame = 0;
	}

	switch (mode)
	{
	case CPU_MODE:
		lenia_basic_CPU();
		lenia_pixels = pixels2;
		break;
	case GPU_MODE: 
		lenia_basic_GPU(); 
		break;
	//case OPENGL_GPU_MODE: bugsCPU(); break;
	}
}

void idle()
{
	glutPostRedisplay();
}

void draw_pixels_zoomed()
{
	// Calculate the size of each grid cell on the screen
	int cell_size_x = SCREEN_X / width_grid;
	int cell_size_y = SCREEN_Y / height_grid;

	for (int i = 0; i < height_grid; i++)
	{
		for (int j = 0; j < width_grid; j++)
		{
			float4 color;
			// Get the color of the current pixel in the grid
			if (mode == GPU_MODE && !tab_1_used) {
				color = pixels2[i * width_grid + j];
			}
			else {
				color = lenia_pixels[i * width_grid + j];
			}
			

			// Draw a rectangle representing the grid cell
			glBegin(GL_QUADS);
			glColor4f(color.y, color.y, color.y, color.w);
			glVertex2i(j * cell_size_x, i * cell_size_y); // Top-left corner
			glVertex2i((j + 1) * cell_size_x, i * cell_size_y); // Top-right corner
			glVertex2i((j + 1) * cell_size_x, (i + 1) * cell_size_y); // Bottom-right corner
			glVertex2i(j * cell_size_x, (i + 1) * cell_size_y); // Bottom-left corner
			glEnd();
		}
	}
}


void render()
{
	calculate();

	//glDrawPixels(SCREEN_X, SCREEN_Y, GL_RGBA, GL_FLOAT, lenia_pixels);
	draw_pixels_zoomed();

	tab_1_used = !tab_1_used;

	glutSwapBuffers();
}

void clean()
{
	switch (mode)
	{
	case CPU_MODE: cleanCPU(); break;
	case GPU_MODE: cleanGPU(); break;
	case OPENGL_GPU_MODE: cleanCPU(); break;
	}

}

void init()
{
	tab_1_used = true;

	switch (mode)
	{
	case CPU_MODE: initCPU(); break;
	case GPU_MODE: initGPU(); break;
	case OPENGL_GPU_MODE: initCPU(); break;
	}
}

void toggleMode(int m)
{
	clean();
	mode = m;
	init();
}

void mouse(int button, int state, int x, int y)
{
	if (button <= 2)
	{
		mx = (float)(scale * (x - SCREEN_X / 2));
		my = -(float)(scale * (y - SCREEN_Y / 2));
	}
	// Wheel reports as button 3 (scroll up) and button 4 (scroll down)
	if (button == 3) scale /= 1.05f;
	else if (button == 4) scale *= 1.05f;
}

void mouseMotion(int x, int y)
{
	mx = (float)(scale * (x - SCREEN_X / 2));
	my = -(float)(scale * (y - SCREEN_Y / 2));
}

void processNormalKeys(unsigned char key, int x, int y) {

	if (key == 27) { clean(); exit(0); }
	else if (key == '1') toggleMode(CPU_MODE);
	else if (key == '2') toggleMode(GPU_MODE);
	//else if (key == '3') toggleMode(OPENGL_GPU_MODE);
}

void processSpecialKeys(int key, int x, int y) {
	// other keys (F1, F2, arrows, home, etc.)
	switch (key) {
	case GLUT_KEY_UP:
		my += 1.0f; // Move the camera up
		break;
	case GLUT_KEY_DOWN:
		my -= 1.0f; // Move the camera down
		break;
	case GLUT_KEY_LEFT:
		mx -= 1.0f; // Move the camera left
		break;
	case GLUT_KEY_RIGHT:
		mx += 1.0f; // Move the camera right
		break;
	}
}

void initGL(int argc, char** argv)
{
	// init GLUT and create window
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(0, 0);
	glutInitWindowSize(SCREEN_X, SCREEN_Y);
	glutCreateWindow(TITLE);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glDisable(GL_DEPTH_TEST);

	// View Ortho
	// Sets up the OpenGL window so that (0,0) corresponds to the top left corner, 
	// and (SCREEN_X,SCREEN_Y) corresponds to the bottom right hand corner.  
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0, SCREEN_X, SCREEN_Y, 0, 0, 1);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.375, 0.375, 0); // Displacement trick for exact pixelization
}


int main(int argc, char** argv) {

	initGL(argc, argv);

	init();

	glutDisplayFunc(render);
	glutIdleFunc(idle);
	glutMotionFunc(mouseMotion);
	glutMouseFunc(mouse);
	glutKeyboardFunc(processNormalKeys);
	glutSpecialFunc(processSpecialKeys);

	GLint GlewInitResult = glewInit();
	if (GlewInitResult != GLEW_OK) {
		printf("ERROR: %s\n", glewGetErrorString(GlewInitResult));
	}

	// enter GLUT event processing cycle
	glutMainLoop();

	clean();

	return 1;
}
